/*
	Mary Barker
	Homework 1

	to compile: nvcc BarkerHW1_Query.cu

OUTPUT (When run on GreyJoy): 

   --- General Information for device 0 ---
	Name: GeForce GTX 580
	Compute capability: 2.0
	Clock rate: 1600000
	Device copy overlap: Enabled
	Kernel execution timeout : Enabled
   --- Memory Information for device 0 ---
	Total global mem: 1542324224
	Total constant mem: 65536
	Max mem pitch: 2147483647
	Texture Alignment: 512
   --- MP Information for device 0 ---
	Multiprocessor count: 16
	Shared mem per mp: 49152
	Registers per mp: 32768
	Threads in warp: 32
	Max threads per block: 1024
	Max thread dimensions: (1024, 1024, 64)
	Max grid dimensions: (65535, 65535, 65535)

   --- General Information for device 1 ---
	Name: GeForce GTX 580
	Compute capability: 2.0
	Clock rate: 1600000
	Device copy overlap: Enabled
	Kernel execution timeout : Disabled
   --- Memory Information for device 1 ---
	Total global mem: 1545469952
	Total constant mem: 65536
	Max mem pitch: 2147483647
	Texture Alignment: 512
   --- MP Information for device 1 ---
	Multiprocessor count: 16
	Shared mem per mp: 49152
	Registers per mp: 32768
	Threads in warp: 32
	Max threads per block: 1024
	Max thread dimensions: (1024, 1024, 64)
	Max grid dimensions: (65535, 65535, 65535)
*/
#include <stdio.h>

int main(void){
	hipDeviceProp_t prop;

	int count;
	hipGetDeviceCount( &count );
	for(int i = 0; i < count; i++)
	{
		hipGetDeviceProperties( &prop, i );
		printf("   --- General Information for device %d ---\n", i);
		printf( "Name: %s\n", prop.name);
		printf( "Compute capability: %d.%d\n", prop.major, prop.minor);
		printf( "Clock rate: %d\n", prop.clockRate);
		printf( "Device copy overlap: ");
		if(prop.deviceOverlap)
			printf("Enabled\n");
		else
			printf("Disabled\n");
		printf("Kernel execution timeout : ");
		if(prop.kernelExecTimeoutEnabled)
			printf("Enabled\n");
		else
			printf("Disabled\n");

		printf("   --- Memory Information for device %d ---\n", i);
		printf("Total global mem: %ld\n", prop.totalGlobalMem);
		printf("Total constant mem: %ld\n", prop.totalConstMem);
		printf("Max mem pitch: %ld\n", prop.memPitch);
		printf("Texture Alignment: %ld\n", prop.textureAlignment);
		printf("   --- MP Information for device %d ---\n", i);
		printf("Multiprocessor count: %d\n", prop.multiProcessorCount);
		printf("Shared mem per mp: %ld\n", prop.sharedMemPerBlock);
		printf("Registers per mp: %d\n", prop.regsPerBlock);
		printf("Threads in warp: %d\n", prop.warpSize);
		printf("Max threads per block: %d\n", prop.maxThreadsPerBlock);
		printf("Max thread dimensions: (%d, %d, %d)\n", prop.maxThreadsDim[0], prop.maxThreadsDim[1], prop.maxThreadsDim[2]);
		printf("Max grid dimensions: (%d, %d, %d)\n", prop.maxGridSize[0], prop.maxGridSize[1], prop.maxGridSize[2]);
		printf("\n");
	}

}

