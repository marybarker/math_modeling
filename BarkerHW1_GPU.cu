/*
 	Mary Barker
	Homework 1

	Vector addition on GPU
	to compile: nvcc BarkerHW1_GPU.cu

	OUTPUTS: 

	N = 100
	Time in milliseconds= 0.026000000000000
	Last Values are A[99] = 198.000000000000000  B[99] = 99.000000000000000  C[99] = 297.000000000000000

	N = 600
	Time in milliseconds= 0.027000000000000
	Last Values are A[599] = 1198.000000000000000  B[599] = 599.000000000000000  C[599] = 1797.000000000000000

	N = 2000
	Time in milliseconds= 0.035000000000000
	Last Values are A[1999] = 3998.000000000000000  B[1999] = 1999.000000000000000  C[1999] = 5997.000000000000000

*/

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

//Length of vectors to be added.
#define N 100  //if N is greater than dimBlock.x program will break

float *A_CPU, *B_CPU, *C_CPU; //CPU pointers

float *A_GPU, *B_GPU, *C_GPU; //GPU pointers

void AllocateMemory()
{					
	//Allocate Device (GPU) Memory, & allocates the value of the specific pointer/array
	hipMalloc(&A_GPU,N*sizeof(float));
	hipMalloc(&B_GPU,N*sizeof(float));
	hipMalloc(&C_GPU,N*sizeof(float));

	//Allocate Host (CPU) Memory
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));
}

//Loads values into vectors that we will add.
void Innitialize()
{
	int i;
	
	for(i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)2*i;	
		B_CPU[i] = (float)i;
	}
}

//Cleaning up memory after we are finished.
void CleanUp(float *A_CPU,float *B_CPU,float *C_CPU,float *A_GPU,float *B_GPU,float *C_GPU)  //free
{
	free(A_CPU); free(B_CPU); free(C_CPU);
	hipFree(A_GPU); hipFree(B_GPU); hipFree(C_GPU);
}

//This is the kernel. It is the function that will run on the GPU.
//It adds vectors A and B then stores result in vector C
__global__ void Addition(float *A, float *B, float *C, int n)
{

	int id = blockIdx.x;
	
	if(id < n) C[id] = A[id] + B[id];
}

int main()
{
	int i;
	timeval start, end;
	
	//Partitioning off the memory that you will be using.
	AllocateMemory();

	//Loading up values to be added.
	Innitialize();
	
	//Starting the timer
	gettimeofday(&start, NULL);

	//Copy Memory from CPU to GPU		
	hipMemcpyAsync(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpyAsync(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	
	//Calling the Kernel (GPU) function.	
	Addition<<<dim3(N), 1>>>(A_GPU, B_GPU, C_GPU, N);
	
	//Copy Memory from GPU to CPU	
	hipMemcpyAsync(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);

	//Stopping the timer
	gettimeofday(&end, NULL);

	//Calculating the total time used in the addition and converting it to milliseconds.
	float time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	
	//Displaying the time 
	printf("Time in milliseconds= %.15f\n", (time/1000.0));	

	// Displaying vector info you will want to comment out the vector print line when your
	//vector becomes big. This is just to make sure everything is running correctly.	
	for(i = 0; i < N; i++)		
	{		
		//printf("A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", i, A_CPU[i], i, B_CPU[i], i, C_CPU[i]);
	}

	//Displaying the last value of the addition for a check when all vector display has been commented out.
	printf("Last Values are A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", N-1, A_CPU[N-1], N-1, B_CPU[N-1], N-1, C_CPU[N-1]);
	
	//You're done so cleanup your mess.
	CleanUp(A_CPU,B_CPU,C_CPU,A_GPU,B_GPU,C_GPU);	
	
	return(0);
}
