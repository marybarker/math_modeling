#include "hip/hip_runtime.h"
/*
	Mary Barker HW 10 
	Ray Tracing with constant memory

	to compile and run: 
		nvcc Barker9.cu -lm -lGL -lGLU -lglut
		./a.out
*/
#include <GL/glut.h>
#include <stdlib.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define INF 2e10f
#define SPHERES 20
#define rnd( x ) (x * rand() / RAND_MAX) 
#define MIN(x,y) (x< y) ? x : y
#define xmin -50
#define xmax  50
#define ymin -50
#define ymax  50

struct Sphere{
	float r, float g, float b;
	float x, float y, float z;
	float radius;
}
Sphere s[SPHERES];
__constant__ Sphere GPUs[SPHERES];

// arrays to hold pixels
float * pixels = NULL; 
float * GPUpixels = NULL; 

//thread format for screen display
unsigned int window_width = 1024;
unsigned int window_height = 1024;
float stepSizeX = (xmax - xmin)/((float)window_width - 1.0);
float stepSizeY = (ymax - ymin)/((float)window_height - 1.0);
dim3 nthreads = MIN(window_width, 1024);
dim3 nblocks = (window_width*window_height - 1) / nthreads.x + 1;

__device__ float hit(float x, float y, float z, float radius, float ox, float oy, float *n ) {
	float dx = ox - x;
	float dy = oy - y;

	if (dx*dx + dy*dy < radius*radius) {
		float dz = sqrtf( radius * radius - dx*dx - dy*dy );
        	*n = dz / sqrtf( radius * radius );
		return dz + z;
	}
	return -INF;
} 

__global__ void trace_rays(float * pix, float dx, float dy, int nx, int ny) {
	int i = blockDim.x * blockIdx.x + threadIdx.x;
	float xx, yy, maxz=-INF;

	if(i < nx * ny) {

		float rr = 0, gg = 0, bb = 0;

		xx = (xmin + threadIdx.x * dx);
		yy = (ymin +  blockIdx.x * dy);

		for(int j = 0; j < SPHERES; j++){

			float n, t = hit(s[j].x, s[j].y, s[j].z, s[j].radius, xx, yy, &n);

			if(t > maxz){
				rr = n * s[j].r;
				gg = n * s[j].g;
				bb = n * s[j].b;
				maxz = t;
			}
		}
		pix[3*i+0] = rr;
		pix[3*i+1] = gg;
		pix[3*i+2] = bb;
	}
}

void allocate_memory() {
	pixels = (float*)malloc(3*window_width*window_height * sizeof(float));
	hipMalloc(&GPUpixels, 3*window_width*window_height * sizeof(float));

	hipMemcpyToSymbol(HIP_SYMBOL(GPUs), s, SPHERES*sizeof(Sphere));

	for(int i = 0; i < SPHERES; i++){
		s[i].x = 		rnd(100.0f) - 50;
		s[i].y = 		rnd(100.0f) - 50;
		s[i].z = 		rnd(100.0f) - 50;
		s[i].r = 		rnd(1.0f);
		s[i].g = 		rnd(1.0f);
		s[i].b = 		rnd(1.0f);
		s[i].radius =	 	rnd(10.0f) + 2;
	}
}

void display(void) { 
	allocate_memory();

	trace_rays<<<nblocks,nthreads>>>(GPUpixels, stepSizeX, stepSizeY, window_width, window_height);

	hipMemcpy(pixels, GPUpixels, 3*window_width*window_height*sizeof(float), hipMemcpyDeviceToHost);

	glDrawPixels(window_width, window_height, GL_RGB, GL_FLOAT, pixels); 
	glFlush(); 
}

int main(int argc, char** argv) { 
   	glutInit(&argc, argv);
	glutInitDisplayMode(GLUT_RGB | GLUT_SINGLE);
   	glutInitWindowSize(window_width, window_height);
   	glutCreateWindow("Fractals man, fractals.");
   	glutDisplayFunc(display);
   	glutMainLoop();
}

