/*
 	Mary Barker
	Homework 6

	Vector dot product on GPU to compile: nvcc BarkerHW6.cu
*/

#include <hip/hip_runtime.h>
#include <sys/time.h>
#include <stdio.h>

#define N 10000  //if N is greater than dimBlock.x program will break
#define MIN(x,y) (x<y)?x:y

float *A_CPU, *B_CPU, *C_CPU; //CPU pointers
float *A_GPU, *B_GPU, *C_GPU; //GPU pointers

dim3 grid, block;

void AllocateMemory()
{					
	//Allocate Device (GPU) Memory, & allocates the value of the specific pointer/array
	hipMalloc(&A_GPU,N*sizeof(float));
	hipMalloc(&B_GPU,N*sizeof(float));
	hipMalloc(&C_GPU,N*sizeof(float));

	//Allocate Host (CPU) Memory
	A_CPU = (float*)malloc(N*sizeof(float));
	B_CPU = (float*)malloc(N*sizeof(float));
	C_CPU = (float*)malloc(N*sizeof(float));

	block = MIN(1024, N);
	grid = (N > 1024) ? ((N - 1) / block.x + 1) : 1;
}

//Loads values into vectors that we will add.
void Innitialize()
{
	int i;
	for(i = 0; i < N; i++)
	{		
		A_CPU[i] = (float)1;	
		B_CPU[i] = (float)1;
	}
}

//Cleaning up memory after we are finished.
void CleanUp(float *A_CPU,float *B_CPU,float *C_CPU,float *A_GPU,float *B_GPU,float *C_GPU)  //free
{
	free(A_CPU); free(B_CPU); free(C_CPU);
	hipFree(A_GPU); hipFree(B_GPU); hipFree(C_GPU);
}

//This is the kernel. It is the function that will run on the GPU.
//It adds vectors A and B then stores result in vector C
__global__ void DotProduct(float *A, float *B, float *C, int n)
{

	int id = blockDim.x * blockIdx.x + threadIdx.x;
	int odd, offset = blockDim.x * blockIdx.x, new_n = blockDim.x;
	bool not_done_yet = true;
	
	if(id < n) C[id] = A[id] * B[id];

	// 'Fold' the vector in half repeatedly
	while(not_done_yet)
	{
		__syncthreads();
		odd = new_n % 2;
		new_n = new_n / 2;
		if(new_n > 0)
		{
			if(id < (offset + new_n))
			{
				if(id + new_n < n)
				{
					C[id] += C[id+new_n];
					if( (odd > 0) && (id < offset + 1) )
						C[id] += C[id+2*new_n];
				}
			}
		}
		else
		{
			not_done_yet = false;
		}
	}
}

int main()
{
	int i;
	timeval start, end;
	
	//Partitioning off the memory that you will be using.
	AllocateMemory();

	//Loading up values to be added.
	Innitialize();
	
	//Starting the timer
	gettimeofday(&start, NULL);

	//Copy Memory from CPU to GPU		
	hipMemcpy(A_GPU, A_CPU, N*sizeof(float), hipMemcpyHostToDevice);
	hipMemcpy(B_GPU, B_CPU, N*sizeof(float), hipMemcpyHostToDevice);

	//Calling the Kernel (GPU) function.	
	DotProduct<<<grid, block>>>(A_GPU, B_GPU, C_GPU, N);
	
	//Copy Memory from GPU to CPU	
	hipMemcpy(C_CPU, C_GPU, N*sizeof(float), hipMemcpyDeviceToHost);

	if(grid.x > 1)
	{
		for(i = 1; i < grid.x; i++)
		{
			C_CPU[0] += C_CPU[i*block.x];
		}
	}

	//Stopping the timer
	gettimeofday(&end, NULL);

	//Calculating the total time used in the addition and converting it to milliseconds.
	float time = (end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec);
	
	//Displaying the time 
	printf("Time in milliseconds= %.15f\n", (time/1000.0));	

	// Displaying vector info you will want to comment out the vector print line when your
	//vector becomes big. This is just to make sure everything is running correctly.	
	for(i = 0; i < N; i++)		
	{		
		//printf("A[%d] = %.15f  B[%d] = %.15f  C[%d] = %.15f\n", i, A_CPU[i], i, B_CPU[i], i, C_CPU[i]);
	}

	//Displaying the value of the dot product
	printf("Value is %f\n", C_CPU[0]);
	
	//You're done so cleanup your mess.
	CleanUp(A_CPU,B_CPU,C_CPU,A_GPU,B_GPU,C_GPU);	
	
	return(0);
}

